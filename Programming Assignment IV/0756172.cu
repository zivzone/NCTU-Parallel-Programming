/**********************************************************************
 * DESCRIPTION:
 *   Cuda Concurrent Wave Equation - Cuda C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */

// define the value on devices
// use pointer to point them
float *device_values;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
//float total_oldval;
//float total_values;
/*void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}*/
__device__ float do_math(float total_oldval, float total_values)
	{
	   float dtime, c, dx, tau, sqtau;
	
	   dtime = 0.3;
	   c = 1.0;
	   dx = 1.0;
	   tau = (c * dtime / dx);
      sqtau = tau * tau;
      // modefy original one into the version of :
      // given total oldval to do math with total current value to get total new value
		float total_newval;
		total_newval = (2.0 * total_values) - total_oldval + (sqtau *  (-2.0)*total_values);
		return total_newval;
	}
	

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
/* void update()
{
   int i, j;

   // Update values for each time step 
   for (i = 1; i<= nsteps; i++) {
      // Update points along line for this time step 
      for (j = 1; j <= tpoints; j++) {
         // global endpoints 
         if ((j == 1) || (j  == tpoints))
            newval[j] = 0.0;
         else
            do_math(j);
      }

      // Update old values with new values 
      for (j = 1; j <= tpoints; j++) {
         oldval[j] = values[j];
         values[j] = newval[j];
      }
   }
}*/
__global__ void update(float *device_values, int tpoints, int nsteps)
{
	int i, j;
	j = (1+threadIdx.x) + blockIdx.x*32;
	if( j <= tpoints ){
		float total_values = device_values[j];
		float total_oldval = total_values;
		float total_newval;
		for(i=1; i<=nsteps; i++){
         // Update points along line for this time step 
         if((j==1) || (j==tpoints))
         // global endpoints 
				total_newval = 0.0;
			else
            total_newval = do_math(total_oldval, total_values);
         // Update old values with new values 
			total_oldval = total_values;
			total_values = total_newval;
      }
      // put current value on device into device_values array 
		device_values[j] = total_values;
	}
}
	

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
   sscanf(argv[2],"%d",&nsteps);
   // Create a space for cuda memory
   hipMalloc(&device_values, sizeof(float)*(1+tpoints));
	check_param();
	printf("Initializing points on the line...\n");
   init_line();
   hipMemcpy(device_values, values, sizeof(float)*(1+tpoints), hipMemcpyHostToDevice);
	printf("Updating all points for all time steps...\n");
   //update();
   // We allocate the memroy considered block number and threadsperbllock
   // You can take the offical document as reference: 3.2.2. Device Memory
   // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
   int block;
   int threadsPerBlock = 256;
	if(tpoints%32){
      // the part of less than 32 still need to be allocate
		block = 1 + tpoints/32;
	}
	else{
		block = tpoints/32;
   }
   update<<<block, threadsPerBlock>>>(device_values, tpoints, nsteps);
	hipMemcpy(values, device_values, sizeof(float)*(1+tpoints), hipMemcpyDeviceToHost);
	printf("Printing final results...\n");
	printfinal();
   printf("\nDone.\n\n");
   hipFree(device_values);
	
	return 0;
}
